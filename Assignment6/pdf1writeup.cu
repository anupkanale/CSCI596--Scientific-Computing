/*----------------------------------------------------------------------
Program pdf0.c computes a pair distribution function for n atoms
given the 3D coordinates of the atoms.
----------------------------------------------------------------------*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <stdlib.h>

#define NHBIN 2000  // Histogram size

float al[3];        // Simulation box lengths
int n;              // Number of atoms
float *r;           // Atomic position array
FILE *fp;

__constant__ float DALTH[3];
__constant__ int DN;
__constant__ float DDRH;

__device__ float d_SignR(float v,float x) {if (x > 0) return v; else return -v;}


__global__ void gpu_histogram_kernel(float *r,float *nhis) {
	int i,j,a,ih;
	float rij,dr;

	int iBlockBegin = (DN/gridDim.x)*blockIdx.x;
	int iBlockEnd = min((DN/gridDim.x)*(blockIdx.x+1),DN);
	int jBlockBegin = (DN/gridDim.y)*blockIdx.y;
	int jBlockEnd = min((DN/gridDim.y)*(blockIdx.y+1),DN);
	for (i=iBlockBegin+threadIdx.x; i<iBlockEnd; i+=blockDim.x) {
		for (j=jBlockBegin+threadIdx.y; j<jBlockEnd; j+=blockDim.y) {
			if (i<j) {
			// Process (i,j) atom pair
			rij = 0.0;
				  for (a=0; a<3; a++) {
				    dr = r[3*i+a]-r[3*j+a];
				    /* Periodic boundary condition */
				    dr = dr-d_SignR(DALTH[a],dr-DALTH[a])-d_SignR(DALTH[a],dr+DALTH[a]);
				    rij += dr*dr;
				  }
				  rij = sqrt(rij); /* Pair distance */
				  ih = rij/DDRH;
				  // nhis[ih] += 1.0; /* Entry to the histogram */
					atomicAdd(&nhis[ih],1.0);
			} // end if i<j
		} // end for j
	} // end for i
}


/*--------------------------------------------------------------------*/
void histogram() {
/*----------------------------------------------------------------------
Constructs a histogram NHIS for atomic-pair distribution.
----------------------------------------------------------------------*/
  float alth[3];
  float* nhis;  // Histogram array
  float rhmax,drh,density,gr;
	int a,ih;

	float* dev_r; // Atomic positions
	float* dev_nhis; // Histogram

  /* Half the simulation box size */
  for (a=0; a<3; a++) alth[a] = 0.5*al[a];
  /* Max. pair distance RHMAX & histogram bin size DRH */
  rhmax = sqrt(alth[0]*alth[0]+alth[1]*alth[1]+alth[2]*alth[2]);
  drh = rhmax/NHBIN;  // Histogram bin size


  nhis = (float*)malloc(sizeof(float)*NHBIN);
  // for (ih=0; ih<NHBIN; ih++) nhis[ih] = 0.0; // Reset the histogram

	hipMalloc((void**)&dev_r,sizeof(float)*3*n);
	hipMalloc((void**)&dev_nhis,sizeof(float)*NHBIN);
	hipMemcpy(dev_r,r,3*n*sizeof(float),hipMemcpyHostToDevice);
	hipMemset(dev_nhis,0.0,NHBIN*sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(DALTH),alth,sizeof(float)*3,0,hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(DN),&n,sizeof(int),0,hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(DDRH),&drh,sizeof(float),0,hipMemcpyHostToDevice);

	dim3 numBlocks(8,8,1);
	dim3 threads_per_block(16,16,1);
	gpu_histogram_kernel<<<numBlocks,threads_per_block>>>(dev_r,dev_nhis);


	// Compute dev_nhis on GPU: dev_r[] ® dev_nhis[]
	hipMemcpy(nhis,dev_nhis,NHBIN*sizeof(float),hipMemcpyDeviceToHost);

  density = n/(al[0]*al[1]*al[2]);
  /* Print out the histogram */
  fp = fopen("pdf_gpu.d","w");
  for (ih=0; ih<NHBIN; ih++) {
    gr = nhis[ih]/(2*M_PI*pow((ih+0.5)*drh,2)*drh*density*n);
    fprintf(fp,"%e %e\n",(ih+0.5)*drh,gr);
  }
  fclose(fp);
  free(nhis);
}
