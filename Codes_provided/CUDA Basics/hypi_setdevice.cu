#include "hip/hip_runtime.h"
// Hybrid MPI+CUDA computation of Pi
#include <stdio.h>
#include <mpi.h>
#include <hip/hip_runtime.h>

#define NBIN  10000000  // Number of bins
#define NUM_BLOCK   13  // Number of thread blocks
#define NUM_THREAD 192  // Number of threads per block

// Kernel that executes on the CUDA device
__global__ void cal_pi(float *sum,int nbin,float step,float offset,int nthreads,int nblocks) {
	int i;
	float x;
	int idx = blockIdx.x*blockDim.x+threadIdx.x;  // Sequential thread index across the blocks
	for (i=idx; i<nbin; i+=nthreads*nblocks) {  // Interleaved bin assignment to threads
		x = offset+(i+0.5)*step;
		sum[idx] += 4.0/(1.0+x*x);
	}
}

int main(int argc,char **argv) {
	int myid,nproc,nbin,tid;
	float step,offset,pi=0.0,pig;
	dim3 dimGrid(NUM_BLOCK,1,1);  // Grid dimensions (only use 1D)
	dim3 dimBlock(NUM_THREAD,1,1);  // Block dimensions (only use 1D)
	float *sumHost,*sumDev;  // Pointers to host & device arrays
	int dev_used;

	MPI_Init(&argc,&argv);
	MPI_Comm_rank(MPI_COMM_WORLD,&myid);  // My MPI rank
	MPI_Comm_size(MPI_COMM_WORLD,&nproc);  // Number of MPI processes
	nbin = NBIN/nproc;  // Number of bins per MPI process
	step = 1.0/(float)(nbin*nproc);  // Step size with redefined number of bins
	offset = myid*step*nbin;  // Quadrature-point offset

	hipSetDevice(myid%2);
	size_t size = NUM_BLOCK*NUM_THREAD*sizeof(float);  //Array memory size
	sumHost = (float *)malloc(size);  //  Allocate array on host
	hipMalloc((void **) &sumDev,size);  // Allocate array on device
	hipMemset(sumDev,0,size);  // Reset array in device to 0
	// Calculate on device (call CUDA kernel)
	cal_pi <<<dimGrid,dimBlock>>> (sumDev,nbin,step,offset,NUM_THREAD,NUM_BLOCK);
	// Retrieve result from device and store it in host array
	hipMemcpy(sumHost,sumDev,size,hipMemcpyDeviceToHost);
	// Reduction over CUDA threads
	for(tid=0; tid<NUM_THREAD*NUM_BLOCK; tid++)
		pi += sumHost[tid];
	pi *= step;
	// CUDA cleanup
	free(sumHost);
	hipFree(sumDev);
	hipGetDevice(&dev_used);
	printf("myid = %d: device used = %d; partial pi = %f\n",myid,dev_used,pi);
	// Reduction over MPI processes
	MPI_Allreduce(&pi,&pig,1,MPI_FLOAT,MPI_SUM,MPI_COMM_WORLD);
	if (myid==0) printf("PI = %f\n",pig);

	MPI_Finalize();
	return 0;
}
